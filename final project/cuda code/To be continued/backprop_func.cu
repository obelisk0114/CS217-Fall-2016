#include "hip/hip_runtime.h"


#define BLOCK_SIZE 512

__global__ void bpnn_layerforward (float *l1, float *l2, float **conn, int n1, int n2) {

	/* Use extern shared memory to store **conn */
	// need to convert 2D array to 1D
	extern __shared__ float temp_conn[];
	float sum;
	int j, k;

	/*** Set up thresholding unit ***/
	l1[0] = 1.0;

	/*** For each unit in second layer ***/
	if (threadIdx.x <= n2 && threadIdx.x != 0) {
		
		/*** Compute weighted sum of its inputs ***/
		sum = 0.0;
		for (k = 0; k <= n1; k++) {
			sum += temp_conn[k * n2 + threadIdx.x] * l1[k];
		}

		l2[threadIdx.x] = squash(sum);
	}
}
